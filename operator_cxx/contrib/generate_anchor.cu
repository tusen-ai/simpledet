#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file generate_proposal.cu
 * \brief Proposal Operator
 * \author Yanghao Li, Chenxia Han
*/
#include <dmlc/logging.h>
#include <dmlc/parameter.h>
#include <mxnet/operator.h>
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "../tensor/sort_op.h"

#include <map>
#include <vector>
#include <string>
#include <utility>
#include <ctime>
#include <iostream>
#include <fstream>
#include <iterator>

#include "../operator_common.h"
#include "../mshadow_op.h"
#include "./generate_anchor-inl.h"

#define FRCNN_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
} while (0)

namespace mshadow {
namespace cuda {
namespace {
// all_anchors are (h * w * anchor, 4)
// w defines "x" and h defines "y"
// count should be total anchors numbers, h * w * anchors
template<typename DType>
__global__ void AnchorGridKernel(const int count,
                                 const int num_anchors,
                                 const int height,
                                 const int width,
                                 const int feature_stride,
                                 double* all_anchors,
                                 DType* out) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    int a = index % num_anchors;
    int w = (index / num_anchors) % width;
    int h = index / num_anchors / width;

    out[index * 4 + 0] = static_cast<DType>(all_anchors[a * 4 + 0] + w * feature_stride);
    out[index * 4 + 1] = static_cast<DType>(all_anchors[a * 4 + 1] + h * feature_stride);
    out[index * 4 + 2] = static_cast<DType>(all_anchors[a * 4 + 2] + w * feature_stride);
    out[index * 4 + 3] = static_cast<DType>(all_anchors[a * 4 + 3] + h * feature_stride);
  }
}

}  // namespace
}  // namespace cuda
}  // namespace mshadow

namespace mxnet {
namespace op {

template<typename xpu>
class GenAnchorGPUOp : public Operator{
 public:
  explicit GenAnchorGPUOp(GenAnchorParam param) {
    this->param_ = param;
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_states) {
    using namespace mshadow;
    using namespace mshadow::expr;
    using namespace mshadow::cuda;

    CHECK_EQ(in_data.size(), 1);
    CHECK_EQ(out_data.size(), 1);
    CHECK_EQ(req.size(), 1);
    CHECK_EQ(req[gen_anchor::kOut], kWriteTo);

    Stream<xpu> *s = ctx.get_stream<xpu>();
    // batch_idx, anchor_idx, height_idx, width_idx
    Tensor<xpu, 4> scores = in_data[gen_anchor::kClsProb].get<xpu, 4, float>(s);
    // height * width * anchors, 4(x1, y1, x2, y2)
    Tensor<xpu, 2> out = out_data[gen_anchor::kOut].get<xpu, 2, float>(s);

    std::vector<double> scales(param_.scales.begin(), param_.scales.end());
    std::vector<double> ratios(param_.ratios.begin(), param_.ratios.end());

    int num_anchors = scales.size() * ratios.size();
    int height = scores.size(2);
    int width = scores.size(3);

    // Generate first anchors based on base anchor
    std::vector<double> base_anchor({
      0.0f, 0.0f, param_.feature_stride - 1.0f, param_.feature_stride - 1.0f
    });
    std::vector<double> anchors;
    gen_anchor_utils::GenerateAnchors(
      base_anchor, ratios, scales, anchors
    );

    // cast to fp32 during AnchorGrid to keep consistency with python implementation
    TensorContainer<gpu, 2, double> out_fp64(out.shape_);

    FRCNN_CUDA_CHECK(
      hipMemcpy(
        out_fp64.dptr_,
        anchors.data(),
        sizeof(decltype(anchors)::value_type) * anchors.size(),
        hipMemcpyHostToDevice
      )
    ); // less than 64K

    /* copy proposals to a mesh grid */
    dim3 dimGrid((out.size(0) + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock);
    dim3 dimBlock(kMaxThreadsPerBlock);
    CheckLaunchParam(dimGrid, dimBlock, "AnchorGrid");
    AnchorGridKernel<<<dimGrid, dimBlock>>>(
      out_fp64.size(0), num_anchors, height, width, param_.feature_stride,
      out_fp64.dptr_, out.dptr_);
    FRCNN_CUDA_CHECK(hipPeekAtLastError());
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_states) {
    using namespace mshadow;
    using namespace mshadow::expr;
    CHECK_EQ(in_grad.size(), 1);

    Stream<xpu> *s = ctx.get_stream<xpu>();
    Tensor<xpu, 4> gscores = in_grad[gen_anchor::kClsProb].get<xpu, 4, float>(s);

    // can not assume the grad would be zero
    Assign(gscores, req[gen_anchor::kClsProb], 0);
  }

 private:
  GenAnchorParam param_;
};  // class GenAnchorGPUOp

template<>
Operator* CreateOp<gpu>(GenAnchorParam param) {
  return new GenAnchorGPUOp<gpu>(param);
}
}  // namespace op
}  // namespace mxnet
