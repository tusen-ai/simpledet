#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file proposal.cu
 * \brief Proposal Operator
 * \author Shaoqing Ren, Jian Guo, Pengfei Chen, Yuntao Chen
*/
#include <dmlc/logging.h>
#include <dmlc/parameter.h>
#include <mxnet/operator.h>
#include <mshadow/tensor.h>
#include <mshadow/cuda/reduce.cuh>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "../tensor/sort_op.h"

#include <map>
#include <vector>
#include <string>
#include <utility>
#include <ctime>
#include <iostream>
#include <fstream>
#include <iterator>

#include "../operator_common.h"
#include "../mshadow_op.h"
#include "./proposal-inl.h"

#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

#define FRCNN_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
} while (0)

namespace mshadow {
namespace cuda {
namespace {
// scores are (b, anchor, h, w)
// proposals are (h * w * anchor, 5)
// w defines "x" and h defines "y"
// count should be total anchors numbers, h * w * anchors
template<typename Dtype>
__global__ void ProposalGridKernel(const int count,
                                   const int num_anchors,
                                   const int height,
                                   const int width,
                                   const int feature_stride,
                                   const Dtype* scores,
                                   Dtype* proposals) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    int a = index % num_anchors;
    int w = (index / num_anchors) % width;
    int h = index / num_anchors / width;

    proposals[index * 5 + 0] = proposals[a * 5 + 0] + w * feature_stride;
    proposals[index * 5 + 1] = proposals[a * 5 + 1] + h * feature_stride;
    proposals[index * 5 + 2] = proposals[a * 5 + 2] + w * feature_stride;
    proposals[index * 5 + 3] = proposals[a * 5 + 3] + h * feature_stride;
    proposals[index * 5 + 4] = scores[(a * height + h) * width + w];
  }
}

// boxes are (h * w * anchor, 5)
// deltas are (b, 4 * anchor, h, w)
// out_pred_boxes are (h * w * anchor, 5)
// count should be total anchors numbers, h * w * anchors
// in-place write: boxes and out_pred_boxes are the same location
template<typename Dtype>
__global__ void BBoxPredKernel(const int count,
                               const int num_anchors,
                               const int feat_height,
                               const int feat_width,
                               const int real_height,
                               const int real_width,
                               const float im_height,
                               const float im_width,
                               const Dtype* boxes,
                               const Dtype* deltas,
                               Dtype* out_pred_boxes) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    int a = index % num_anchors;
    int w = (index / num_anchors) % feat_width;
    int h = index / num_anchors / feat_width;

    float width = boxes[index * 5 + 2] - boxes[index * 5 + 0] + 1.0f;
    float height = boxes[index * 5 + 3] - boxes[index * 5 + 1] + 1.0f;
    float ctr_x = boxes[index * 5 + 0] + 0.5f * (width - 1.0f);
    float ctr_y = boxes[index * 5 + 1] + 0.5f * (height - 1.0f);

    float dx = deltas[((a * 4) * feat_height + h) * feat_width + w];
    float dy = deltas[((a * 4 + 1) * feat_height + h) * feat_width + w];
    float dw = deltas[((a * 4 + 2) * feat_height + h) * feat_width + w];
    float dh = deltas[((a * 4 + 3) * feat_height + h) * feat_width + w];

    float pred_ctr_x = dx * width + ctr_x;
    float pred_ctr_y = dy * height + ctr_y;
    float pred_w = exp(dw) * width;
    float pred_h = exp(dh) * height;

    float pred_x1 = pred_ctr_x - 0.5f * (pred_w - 1.0f);
    float pred_y1 = pred_ctr_y - 0.5f * (pred_h - 1.0f);
    float pred_x2 = pred_ctr_x + 0.5f * (pred_w - 1.0f);
    float pred_y2 = pred_ctr_y + 0.5f * (pred_h - 1.0f);

    pred_x1 = max(min(pred_x1, im_width - 1.0f), 0.0f);
    pred_y1 = max(min(pred_y1, im_height - 1.0f), 0.0f);
    pred_x2 = max(min(pred_x2, im_width - 1.0f), 0.0f);
    pred_y2 = max(min(pred_y2, im_height - 1.0f), 0.0f);

    out_pred_boxes[index * 5 + 0] = pred_x1;
    out_pred_boxes[index * 5 + 1] = pred_y1;
    out_pred_boxes[index * 5 + 2] = pred_x2;
    out_pred_boxes[index * 5 + 3] = pred_y2;

    if (h >= real_height || w >= real_width) {
      out_pred_boxes[index * 5 + 4] = -1.0f;
    }
  }
}

// boxes are (h * w * anchor, 5)
// deltas are (b, 4 * anchor, h, w)
// out_pred_boxes are (h * w * anchor, 5)
// count should be total anchors numbers, h * w * anchors
// in-place write: boxes and out_pred_boxes are the same location
template<typename Dtype>
__global__ void IoUPredKernel(const int count,
                              const int num_anchors,
                              const int feat_height,
                              const int feat_width,
                              const int real_height,
                              const int real_width,
                              const float im_height,
                              const float im_width,
                              const Dtype* boxes,
                              const Dtype* deltas,
                              Dtype* out_pred_boxes) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    int a = index % num_anchors;
    int w = (index / num_anchors) % feat_width;
    int h = index / num_anchors / feat_width;

    float x1 = boxes[index * 5 + 0];
    float y1 = boxes[index * 5 + 1];
    float x2 = boxes[index * 5 + 2];
    float y2 = boxes[index * 5 + 3];

    float dx1 = deltas[((a * 4) * feat_height + h) * feat_width + w];
    float dy1 = deltas[((a * 4 + 1) * feat_height + h) * feat_width + w];
    float dx2 = deltas[((a * 4 + 2) * feat_height + h) * feat_width + w];
    float dy2 = deltas[((a * 4 + 3) * feat_height + h) * feat_width + w];

    float pred_x1 = max(min(x1 + dx1, im_width - 1.0f), 0.0f);
    float pred_y1 = max(min(y1 + dy1, im_height - 1.0f), 0.0f);
    float pred_x2 = max(min(x2 + dx2, im_width - 1.0f), 0.0f);
    float pred_y2 = max(min(y2 + dy2, im_height - 1.0f), 0.0f);

    out_pred_boxes[index * 5 + 0] = pred_x1;
    out_pred_boxes[index * 5 + 1] = pred_y1;
    out_pred_boxes[index * 5 + 2] = pred_x2;
    out_pred_boxes[index * 5 + 3] = pred_y2;

    if (h >= real_height || w >= real_width) {
      out_pred_boxes[index * 5 + 4] = -1.0f;
    }
  }
}

// filter box with stride less than rpn_min_size
// filter: set score to zero
// dets (n, 5)
template<typename Dtype>
__global__ void FilterBoxKernel(const int count,
                                const float min_size,
                                Dtype* dets) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    float iw = dets[index * 5 + 2] - dets[index * 5 + 0] + 1.0f;
    float ih = dets[index * 5 + 3] - dets[index * 5 + 1] + 1.0f;
    if (iw < min_size || ih < min_size) {
      dets[index * 5 + 0] -= min_size / 2;
      dets[index * 5 + 1] -= min_size / 2;
      dets[index * 5 + 2] += min_size / 2;
      dets[index * 5 + 3] += min_size / 2;
      dets[index * 5 + 4] = -1.0f;
    }
  }
}

// copy score and init order
// dets (n, 5); score (n, ); order (n, )
// count should be n (total anchors or proposals)
template<typename Dtype>
__global__ void CopyScoreKernel(const int count,
                                const Dtype* dets,
                                Dtype* score,
                                int* order) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    score[index] = dets[index * 5 + 4];
    order[index] = index;
  }
}

// reorder proposals according to order and keep the top_n proposals
// prev_dets (n, 5); order (n, ); dets (n, 5)
// count should be output anchor numbers (top_n)
template<typename Dtype>
__global__ void ReorderProposalsKernel(const int count,
                                       const Dtype* prev_dets,
                                       const int* order,
                                       Dtype* dets) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    const int order_i = order[index];
    for (int j = 0; j < 5; j ++) {
      dets[index * 5 + j] = prev_dets[order_i * 5 + j];
    }
  }
}

__device__ inline float devIoU(float const * const a, float const * const b) {
  float left = max(a[0], b[0]), right = min(a[2], b[2]);
  float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
  float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
  float interS = width * height;
  float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
  float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
  return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thresh,
                           const float *dev_boxes, uint64_t *dev_mask) {
  const int threadsPerBlock = sizeof(uint64_t) * 8;
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ float block_boxes[threadsPerBlock * 5];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 5 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 0];
    block_boxes[threadIdx.x * 5 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 1];
    block_boxes[threadIdx.x * 5 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 2];
    block_boxes[threadIdx.x * 5 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 3];
    block_boxes[threadIdx.x * 5 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + 4];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const float *cur_box = dev_boxes + cur_box_idx * 5;
    int i = 0;
    uint64_t t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

void _nms(const mshadow::Tensor<gpu, 2>& boxes,
          const float nms_overlap_thresh,
          int *keep,
          int *num_out,
          uint64_t *mask_dev,
          uint64_t *mask_host) {
  /*
  @input  boxes: (pre_nms_top_n, 5)
  @return keep
  @return num_out
  @tmp    mask_dev
  @tmp    mask_host
  */
  const int threadsPerBlock = sizeof(uint64_t) * 8;
  const int boxes_num = boxes.size(0);
  const int boxes_dim = boxes.size(1);

  float* boxes_dev = boxes.dptr_;

  const int col_blocks = DIVUP(boxes_num, threadsPerBlock);

  dim3 blocks(DIVUP(boxes_num, threadsPerBlock),
              DIVUP(boxes_num, threadsPerBlock));
  dim3 threads(threadsPerBlock);
  nms_kernel<<<blocks, threads>>>(boxes_num,
                                  nms_overlap_thresh,
                                  boxes_dev,
                                  mask_dev);
  FRCNN_CUDA_CHECK(hipPeekAtLastError());

  // TODO: need to be rewritten
  FRCNN_CUDA_CHECK(hipMemcpy(mask_host,
                              mask_dev,
                              sizeof(uint64_t) * boxes_num * col_blocks,
                              hipMemcpyDeviceToHost));

  std::vector<uint64_t> remv(col_blocks);
  memset(&remv[0], 0, sizeof(uint64_t) * col_blocks);

  int num_to_keep = 0;
  for (int i = 0; i < boxes_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep[num_to_keep++] = i;
      uint64_t *p = mask_host + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }
  *num_out = num_to_keep;
}

// copy proposals to output
// dets (top_n, 5); keep (top_n, ); out (top_n, )
// count should be top_n (total anchors or proposals)
template<typename Dtype>
__global__ void PrepareOutput(const int count,
                              const Dtype* dets,
                              const int* keep,
                              const int out_size,
                              Dtype* out,
                              Dtype* score,
                              const bool is_train) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < count;
       index += blockDim.x * gridDim.x) {
    if (index < out_size) {
      int keep_i = keep[index];
      for (int j = 0; j < 4; ++j) {
        out[index * 4 + j] = dets[keep_i * 5 + j];
      }
      score[index] = dets[keep_i * 5 + 4];
    } else {
      int keep_i = keep[index % out_size];
      for (int j = 0; j < 4; ++j) {
          if (is_train) {
            out[index * 4 + j] = dets[keep_i * 5 + j];
          } else {
            out[index * 4 + j] = 0.0f;
          }
      }
      if (is_train) {
        score[index] = dets[keep_i * 5 + 4];
      } else {
        score[index] = 0;
      }
    }
  }
}

}  // namespace
}  // namespace cuda
}  // namespace mshadow

namespace mxnet {
namespace op {

template<typename xpu>
class ProposalGPUOp : public Operator{
 public:
  explicit ProposalGPUOp(ProposalParam param) {
    this->param_ = param;
  }

  virtual void Forward(const OpContext &ctx,
                       const std::vector<TBlob> &in_data,
                       const std::vector<OpReqType> &req,
                       const std::vector<TBlob> &out_data,
                       const std::vector<TBlob> &aux_states) {
    using namespace mshadow;
    using namespace mshadow::expr;
    using namespace mshadow::cuda;

    CHECK_EQ(in_data.size(), 3);
    CHECK_EQ(out_data.size(), 2);
    CHECK_GT(req.size(), 1);
    // CHECK_EQ(req[proposal::kOut], kWriteTo);

    Stream<xpu> *s = ctx.get_stream<xpu>();

    Tensor<xpu, 4> scores = in_data[proposal::kClsProb].get<xpu, 4, float>(s); // batch_idx, anchor_idx, height_idx, width_idx
    Tensor<xpu, 4> bbox_deltas = in_data[proposal::kBBoxPred].get<xpu, 4, float>(s); // batch_idx, height_idx, width_idx, anchor_idx
    Tensor<xpu, 2> im_info = in_data[proposal::kImInfo].get<xpu, 2, float>(s); // batch_idx, 3(height, width, scale)

    Tensor<xpu, 3> out = out_data[proposal::kOut].get<xpu, 3, float>(s); // batch_idx, rois_idx, 4(x1, y1, x2, y2), batch_idx is needed after flatten
    Tensor<xpu, 3> out_score = out_data[proposal::kScore].get<xpu, 3, float>(s); // batch_idx, rois_idx, 1(score)

    uint64_t WORKSPACE_LIMIT = 1024 * 1024 * param_.workspace; // 256 MB should be sufficient
    Tensor<xpu, 1, uint8_t> workspace = ctx.requested[proposal::kTempSpace].get_space_typed<xpu, 1, uint8_t>(Shape1(WORKSPACE_LIMIT), s);
    uint64_t allocated_bytes = 0ULL;
    uint64_t allocated_bytes_outside_loop = 0ULL;

    int nbatch = scores.size(0);
    int num_anchors = scores.size(1) / 2;
    int height = scores.size(2);
    int width = scores.size(3);
    int count = num_anchors * height * width;  // count of total anchors
    // set to -1 for max
    int rpn_pre_nms_top_n = (param_.rpn_pre_nms_top_n > 0) ? param_.rpn_pre_nms_top_n : count;
    rpn_pre_nms_top_n = std::min(rpn_pre_nms_top_n, count);
    int rpn_post_nms_top_n = std::min(param_.rpn_post_nms_top_n, rpn_pre_nms_top_n);
    if (!param_.is_train) {
        rpn_post_nms_top_n = param_.rpn_post_nms_top_n;
    }

    // Generate first anchors based on base anchor
    std::vector<float> base_anchor(4);
    base_anchor[0] = 0.0;
    base_anchor[1] = 0.0;
    base_anchor[2] = param_.feature_stride - 1.0;
    base_anchor[3] = param_.feature_stride - 1.0;
    CHECK_EQ(num_anchors, param_.ratios.info.size() * param_.scales.info.size());
    std::vector<float> anchors;
    proposal_utils::GenerateAnchors(base_anchor,
                           param_.ratios.info,
                           param_.scales.info,
                           &anchors);

    // Copy generated anchors to GPU
    Tensor<xpu, 3> proposals(reinterpret_cast<float *>(workspace.dptr_ + allocated_bytes), Shape3(nbatch, count, 5));
    allocated_bytes += nbatch * count * 5 * sizeof(float);
    CHECK_LT(allocated_bytes, WORKSPACE_LIMIT) << "Allocating more memory than workspace limit";

    // im_info is small, we want to copy them to cpu
    std::vector<float> cpu_im_info(nbatch * 3);
    FRCNN_CUDA_CHECK(hipMemcpy(cpu_im_info.data(), 
                                im_info.dptr_,
                                sizeof(float) * cpu_im_info.size(),
                                hipMemcpyDeviceToHost)); // less than 64K

    
    Shape<3> fg_scores_shape = Shape3(in_data[proposal::kClsProb].shape_[1] / 2,
                                      in_data[proposal::kClsProb].shape_[2], 
                                      in_data[proposal::kClsProb].shape_[3]);

    allocated_bytes_outside_loop = allocated_bytes;
    /* copy anchors for all images in batch */
    for (int i = 0; i < nbatch; i++) {
      // prevent padded predictions
      int real_height = static_cast<int>(cpu_im_info[i*3 + 0] / param_.feature_stride);
      int real_width = static_cast<int>(cpu_im_info[i*3 + 1] / param_.feature_stride);
      CHECK_GE(height, real_height) << height << " " << real_height << std::endl;
      CHECK_GE(width, real_width) << width << " " << real_width << std::endl;

      float* batch_proposals = proposals.dptr_ + i * 5 * count;
      FRCNN_CUDA_CHECK(hipMemcpy(batch_proposals,
                                  &anchors[0], 
                                  sizeof(float) * anchors.size(),
                                  hipMemcpyHostToDevice)); // less than 64K

      /* get current batch foreground score */
      float* foreground_score_ptr = reinterpret_cast<float *>(scores.dptr_) + i * 2 * count + fg_scores_shape.Size();
      Tensor<xpu, 3> fg_scores = Tensor<xpu, 3>(foreground_score_ptr, fg_scores_shape);

      /* copy proposals to a mesh grid */
      dim3 dimGrid((count + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock);
      dim3 dimBlock(kMaxThreadsPerBlock);
      CheckLaunchParam(dimGrid, dimBlock, "ProposalGrid");
      ProposalGridKernel<<<dimGrid, dimBlock>>>(
        count, num_anchors, height, width, param_.feature_stride,
        fg_scores.dptr_, batch_proposals);
      FRCNN_CUDA_CHECK(hipPeekAtLastError());

      /* transform anchors and bbox_deltas into bboxes */
      CheckLaunchParam(dimGrid, dimBlock, "BBoxPred");
      if (param_.iou_loss) {
        IoUPredKernel<<<dimGrid, dimBlock>>>(
          count, num_anchors, height, width, real_height, real_width,
          cpu_im_info[i * 3 + 0], cpu_im_info[i * 3 + 1],
          batch_proposals, bbox_deltas.dptr_ + i * 4 * count, batch_proposals);
      } else {
        BBoxPredKernel<<<dimGrid, dimBlock>>>(
          count, num_anchors, height, width, real_height, real_width,
          cpu_im_info[i * 3 + 0], cpu_im_info[i * 3 + 1],
          batch_proposals, bbox_deltas.dptr_ + i * 4 * count, batch_proposals);
      }
      FRCNN_CUDA_CHECK(hipPeekAtLastError());

      /* filter boxes with less than rpn_min_size */
      CheckLaunchParam(dimGrid, dimBlock, "FilterBox");
      FilterBoxKernel<<<dimGrid, dimBlock>>>(
        count, param_.rpn_min_size * cpu_im_info[i * 3 + 2], batch_proposals);
      FRCNN_CUDA_CHECK(hipPeekAtLastError());

      /* copy score to a continuous memory */
      Tensor<xpu, 1> score(reinterpret_cast<float *>(workspace.dptr_ + allocated_bytes), Shape1(count));
      allocated_bytes += count * sizeof(float);
      CHECK_LT(allocated_bytes, WORKSPACE_LIMIT) << "Allocating more memory than workspace limit";      

      Tensor<xpu, 1, int> order(reinterpret_cast<int *>(workspace.dptr_ + allocated_bytes), Shape1(count));
      allocated_bytes += count * sizeof(int);
      CHECK_LT(allocated_bytes, WORKSPACE_LIMIT) << "Allocating more memory than workspace limit";      

      CheckLaunchParam(dimGrid, dimBlock, "CopyScore");
      CopyScoreKernel<<<dimGrid, dimBlock>>>(
        count, batch_proposals, score.dptr_, order.dptr_);
      FRCNN_CUDA_CHECK(hipPeekAtLastError());

      /* argsort score, save order */
      thrust::stable_sort_by_key(thrust::device,
                                 score.dptr_,
                                 score.dptr_ + score.size(0),
                                 order.dptr_,
                                 thrust::greater<float>());
      FRCNN_CUDA_CHECK(hipPeekAtLastError());

      /* Reorder proposals according to order */
      Tensor<xpu, 2> ordered_proposals(reinterpret_cast<float *>(workspace.dptr_ + allocated_bytes), Shape2(rpn_pre_nms_top_n, 5));
      allocated_bytes += rpn_pre_nms_top_n * 5 * sizeof(float);
      CHECK_LT(allocated_bytes, WORKSPACE_LIMIT) << "Allocating more memory than workspace limit";      

      dimGrid.x = (rpn_pre_nms_top_n + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
      CheckLaunchParam(dimGrid, dimBlock, "ReorderProposals");
      ReorderProposalsKernel<<<dimGrid, dimBlock>>>(
        rpn_pre_nms_top_n, batch_proposals, order.dptr_, ordered_proposals.dptr_);
      FRCNN_CUDA_CHECK(hipPeekAtLastError());

      /* perform nms */
      std::vector<int> _keep(rpn_pre_nms_top_n);
      int out_size = 0;
      const int boxes_num = rpn_pre_nms_top_n;
      const int col_blocks = DIVUP(boxes_num, sizeof(uint64_t) * 8);
      // take special care when allocate memory of 8-byte alignment.
      allocated_bytes += allocated_bytes % sizeof(uint64_t);
      Tensor<xpu, 1, uint64_t> mask_tensor(reinterpret_cast<uint64_t *>(workspace.dptr_ + allocated_bytes), Shape1(boxes_num * col_blocks));
      allocated_bytes += boxes_num * col_blocks * sizeof(uint64_t); 
      CHECK_LT(allocated_bytes, WORKSPACE_LIMIT) << "Allocating more memory than workspace limit";   
      // the following line does not need change since it the only place where requires host workspace
      Tensor<cpu, 1, uint64_t> mask_host_tensor = ctx.requested[proposal::kTempSpace].get_host_space_typed<1, uint64_t>(Shape1(boxes_num * col_blocks));
      uint64_t *mask_dev = mask_tensor.dptr_;
      uint64_t *mask_host = mask_host_tensor.dptr_;
      _nms(ordered_proposals,
           param_.threshold,
           &_keep[0],
           &out_size,
           mask_dev,
           mask_host);

      /* copy nms result to gpu */
      Tensor<xpu, 1, int> keep(reinterpret_cast<int *>(workspace.dptr_ + allocated_bytes), Shape1(_keep.size()));
      allocated_bytes += _keep.size() * sizeof(int);
      CHECK_LT(allocated_bytes, WORKSPACE_LIMIT) << "Allocating more memory than workspace limit";
      
      FRCNN_CUDA_CHECK(hipMemcpy(keep.dptr_, 
                                  &_keep[0], 
                                  sizeof(int) * _keep.size(),
                                  hipMemcpyHostToDevice)); // less than 64K

      /* copy results after nms */
      dimGrid.x = (rpn_post_nms_top_n + kMaxThreadsPerBlock - 1) / kMaxThreadsPerBlock;
      CheckLaunchParam(dimGrid, dimBlock, "PrepareOutput");
      PrepareOutput<<<dimGrid, dimBlock>>>(
        rpn_post_nms_top_n, ordered_proposals.dptr_, keep.dptr_, out_size,
        out.dptr_ + i * 4 * rpn_post_nms_top_n,
        out_score.dptr_ + i * rpn_post_nms_top_n,
        param_.is_train);
      FRCNN_CUDA_CHECK(hipPeekAtLastError());
      
      // recycle all bytes allocated within loop
      allocated_bytes = allocated_bytes_outside_loop;
    }
  }

  virtual void Backward(const OpContext &ctx,
                        const std::vector<TBlob> &out_grad,
                        const std::vector<TBlob> &in_data,
                        const std::vector<TBlob> &out_data,
                        const std::vector<OpReqType> &req,
                        const std::vector<TBlob> &in_grad,
                        const std::vector<TBlob> &aux_states) {
    using namespace mshadow;
    using namespace mshadow::expr;
    CHECK_EQ(in_grad.size(), 3);

    Stream<xpu> *s = ctx.get_stream<xpu>();
    Tensor<xpu, 4> gscores = in_grad[proposal::kClsProb].get<xpu, 4, real_t>(s);
    Tensor<xpu, 4> gbbox = in_grad[proposal::kBBoxPred].get<xpu, 4, real_t>(s);
    Tensor<xpu, 2> ginfo = in_grad[proposal::kImInfo].get<xpu, 2, real_t>(s);

    // can not assume the grad would be zero
    Assign(gscores, req[proposal::kClsProb], 0);
    Assign(gbbox, req[proposal::kBBoxPred], 0);
    Assign(ginfo, req[proposal::kImInfo], 0);
  }

 private:
  ProposalParam param_;
};  // class ProposalGPUOp

template<>
Operator* CreateOp<gpu>(ProposalParam param) {
  return new ProposalGPUOp<gpu>(param);
}
}  // namespace op
}  // namespace mxnet
